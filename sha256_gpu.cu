#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <stdbool.h>
#include <stdint.h>
#include <stdlib.h>

extern "C" {
#include "sha256.h"
#include "utils.h"
}

#define DATASIZE 55

__constant__ static const WORD k[64] = {
  0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
  0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
  0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
  0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
  0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
  0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
  0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
  0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

__constant__ static const WORD sha256init[8] = {
    0x6a09e667,0xbb67ae85,0x3c6ef372,0xa54ff53a,0x510e527f,0x9b05688c,0x1f83d9ab,0x5be0cd19
};


__global__ void kernel_sha256(BYTE *data, unsigned int* difficulty, Nonce_result *nr,unsigned int *multiplier, uint64_t* snonce, WORD* m1, WORD* l1, WORD* nonceRounds);
__device__ WORD hash2int(BYTE h[32]);

inline void gpuAssert(hipError_t code, char *file, int line, bool abort)
{
    if (code != hipSuccess)
    {
	fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
	if (abort) exit(code);
    }
}

#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__, true); }

extern "C" bool amoveo_mine_gpu(BYTE nonce[23],unsigned int difficulty,BYTE data[55],unsigned int GDIM, unsigned int BDIM, unsigned int multiplier,unsigned int nonceRounds,double* numHashes)
{
    //Host Side Preprocessing

    //ASSUME that first 12 bytes of nonce (data bytes 32-43) are 0
    //ASSSME that next 4 bytes of nonce (data bytes 44-47) are random (given from server in data) for nonce space expansion
    //ASSUME that last 7 bytes of nonce (data bytes 48-54) is nonce space explored (by kernel + multiplier (5 bytes) + noncerounds (2 bytes))
    for(int i = 32; i < 44; i++)
    {
	data[i] = 0x00;
    }
    
    //Initialize Cuda Grid variables
    dim3 DimGrid(GDIM,GDIM);
    dim3 DimBlock(BDIM,1);
    
    //Used to store a nonce if a block is mined
    Nonce_result h_nr;
    initialize_nonce_result(&h_nr);

    //DANGER!!! Beware of 32-bit overflow when multiplying before assigning
    uint64_t s_nonce = GDIM*GDIM; 
    s_nonce *= BDIM;
    s_nonce *= multiplier;

    //Initial shared state
    WORD i,j,t1,t2;
    WORD m1[12];
    #pragma unroll
    for (i = 0, j = 0; i < 8; ++i, j += 4)
	m1[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);

    m1[8] = 0x00000000;
    m1[9] = 0x00000000;
    m1[10] = 0x00000000;
    m1[11] = (data[44] << 24) | (data[45] << 16) | (data[46] << 8) | (data[47]);

    WORD l1[8];
    #pragma unroll
    for(i = 0; i < 8; i++){
	l1[i] = sha256init[i];
    }

    #pragma unroll
    for(i = 0; i < 12; i++){
	t1 = l1[7] + EP1(l1[4]) + CH(l1[4],l1[5],l1[6]) + k[i] + m1[i];
	t2 = EP0(l1[0]) + MAJ(l1[0],l1[1],l1[2]);
	l1[7] = l1[6];
	l1[6] = l1[5];
	l1[5] = l1[4];
	l1[4] = l1[3] + t1;
	l1[3] = l1[2];
	l1[2] = l1[1];
	l1[1] = l1[0];
	l1[0] = t1 + t2;
    }
    
    //Allocate space on Global Memory
    BYTE *d_data;
    unsigned int *d_difficulty;
    Nonce_result *d_nr;
    unsigned int *d_multiplier;
    uint64_t *d_snonce;
    WORD* d_m1;
    WORD* d_l1;
    WORD* d_noncerounds;
 
    CUDA_SAFE_CALL(hipMalloc((void **)&d_data, DATASIZE*sizeof(BYTE)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_difficulty, sizeof(unsigned int)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_nr, sizeof(Nonce_result)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_multiplier, sizeof(unsigned int)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_snonce, sizeof(uint64_t)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_m1, 12*sizeof(WORD)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_l1, 8*sizeof(WORD)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_noncerounds, sizeof(WORD)));
    
    //Copy data to device
    CUDA_SAFE_CALL(hipMemcpy(d_data, (void *) data, DATASIZE*sizeof(BYTE), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_difficulty, (void *) &difficulty, sizeof(unsigned int), hipMemcpyHostToDevice));       
    CUDA_SAFE_CALL(hipMemcpy(d_nr, (void *) &h_nr, sizeof(Nonce_result), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_multiplier, (void *) &multiplier, sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_snonce, (void *) &s_nonce, sizeof(uint64_t),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_m1, (void*) m1, 12*sizeof(WORD),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_l1,(void*) l1, 8*sizeof(WORD),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_noncerounds,(void*) &nonceRounds, sizeof(WORD),hipMemcpyHostToDevice));
    
    kernel_sha256<<<DimGrid, DimBlock>>>(d_data,d_difficulty,d_nr,d_multiplier,d_snonce,d_m1,d_l1,d_noncerounds);
    
    //Copy nonce result back to host
    CUDA_SAFE_CALL(hipMemcpy((void *) &h_nr, d_nr, sizeof(Nonce_result), hipMemcpyDeviceToHost));

    hipDeviceSynchronize();
    
    //Free memory on device
    CUDA_SAFE_CALL(hipFree(d_data));
    CUDA_SAFE_CALL(hipFree(d_difficulty));
    CUDA_SAFE_CALL(hipFree(d_nr));
    CUDA_SAFE_CALL(hipFree(d_multiplier));
    CUDA_SAFE_CALL(hipFree(d_snonce));
    CUDA_SAFE_CALL(hipFree(d_m1));
    CUDA_SAFE_CALL(hipFree(d_l1));
    CUDA_SAFE_CALL(hipFree(d_noncerounds));
    
    if(h_nr.nonce_found){
	for(int i=32; i<55;i++)
	    nonce[i-32]=data[i];	
	for(int i=0; i<sizeof(int64_t)-1; i++)
	    nonce[16+i] = ((BYTE*)(&h_nr.nonce))[i];
    }

    *numHashes = ((double)GDIM)*((double)GDIM)*((double)BDIM)*nonceRounds;
    return h_nr.nonce_found;
}

/**************************** VARIABLES *****************************/
__inline__ __device__ WORD hash2int(WORD letters[8]){
    WORD total_zeros = 0;
    
    #pragma unroll 8
    for(int i=0; i < 8; i++)
    {
	int num_zeros = __clz(sha256init[i]+letters[i]);//ok to use sha256init because it's only 1 round of SHA256
	total_zeros += num_zeros;

	if(num_zeros < 32)
	    break;
    }
  
    int hash_index = total_zeros/8; //Index that we would get if we were working with the little-endian hash[32]
    int byte_zeros = total_zeros-8*hash_index; //Number of zeros in the byte with less than 8 zeros
    
    int state_index_1 = hash_index/4;
    int state_index_2 = (hash_index+1)/4;
    //state is big endian, but we want to work in little endian, so have to reverse byte order of words
    int byte_index_1 = 3-hash_index%4;
    int byte_index_2 = 3-(hash_index+1)%4;
    
    WORD w1 = sha256init[state_index_1]+letters[state_index_1];
    WORD w2 = sha256init[state_index_2]+letters[state_index_2];
    
    WORD y = (((BYTE*)&w2)[byte_index_2])/(1<<(7-byte_zeros));
    if(byte_zeros < 7)
	y += ((((BYTE*)&w1)[byte_index_1])%(1<<(7-byte_zeros)))*(1<<(byte_zeros+1));

    return 256*total_zeros+y;
}

#define NONCE_VAL (gridDim.x*blockDim.x*blockIdx.y + blockDim.x*blockIdx.x + threadIdx.x)

__global__ void kernel_sha256(BYTE *data, unsigned int *difficulty, Nonce_result *nr, unsigned int* multiplier, uint64_t *s_nonce, WORD* m1, WORD* l1, WORD* nonceRounds) {
    
    if(nr->nonce_found) return;
    
    WORD i,j,t1,t2;
    
    uint64_t nonce = *s_nonce+NONCE_VAL;
    BYTE* byte_nonce = (BYTE *)&nonce;

    //Unroll sha256_init
    WORD m2[54];//m for nonce + SHA256 extension (without m of block data + early 0 nonces (m1))
    WORD l2[8]; //SHA-256 letters a-h for nonce + extension (without l of block data - l1)
  
    //////////////////////////////////////////////////////////////////////
    //Fill m2 from sha256_final
    #pragma unroll
    for(WORD n = 0; n < *nonceRounds; n++){
	m2[0] = (byte_nonce[0]<<24) | (byte_nonce[1]<<16) | (byte_nonce[2]<<8) | (byte_nonce[3]);
	m2[1] = (byte_nonce[4]<<24) | (((BYTE*)&n)[0]<<16) | (((BYTE*)&n)[1]<<8) | (0x80);

	m2[2] = 0x00000000;//((440>>56)<<24) | ((440>>48)<<16) | ((440>>40)<<8) | (440>>32);
	m2[3] = 0x000001b8;//((440>>24)<<24) | ((440>>16)<<16) | ((440>>8)<<8) | (440);

        #pragma unroll
	for(i=4; i<7; i++)
	    m2[i] = SIG1(m2[i-2]) + m1[i+5] + SIG0(m1[i-3]) + m1[i-4];
    
        #pragma unroll
	for(i=7; i<15; i++)
	    m2[i] = SIG1(m2[i-2]) + m2[i-7] + SIG0(m1[i-3]) + m1[i-4];
    
	m2[15] = SIG1(m2[13]) + m2[8] + SIG0(m2[0]) + m1[11];
    
        #pragma unroll
	for (i=16 ; i < 52; i++)
	    m2[i] = SIG1(m2[i - 2]) + m2[i - 7] + SIG0(m2[i - 15]) + m2[i - 16];
	///////////////////////////////////////////////////////////////

	///////////////////////////////////////////////////////////////
	//Calculate state from sha256_final
    
        #pragma unroll
	for(i = 0; i < 8; i++){
	    l2[i] = l1[i];
	}
	
        #pragma unroll
	for(i = 12; i < 64; i++){
	    t1 = l2[7] + EP1(l2[4]) + CH(l2[4],l2[5],l2[6]) + k[i] + m2[i-12];
	    t2 = EP0(l2[0]) + MAJ(l2[0],l2[1],l2[2]);
	    l2[7] = l2[6];
	    l2[6] = l2[5];
	    l2[5] = l2[4];
	    l2[4] = l2[3] + t1;
	    l2[3] = l2[2];
	    l2[2] = l2[1];
	    l2[1] = l2[0];
	    l2[0] = t1 + t2;
	}

	//Don't convert from SHA big endian to little endian since __clz in hash2int (below) uses big endian.
	//////////////////////////////////////////////////////////////
    
	if( hash2int(l2) > *difficulty)
	{
	    //DANGER! RACE CONDITION HERE TO WRITE RESULTS!!!
	    //Code below is OK for now because nonce assignment is atomic,
	    //and we don't care which nonce we find.
	    nr->nonce_found = true;
	    byte_nonce[5] = ((BYTE*)&n)[0];
	    byte_nonce[6] = ((BYTE*)&n)[1];
	    nr->nonce = nonce;
	    return;
	}
    }
}
